
#include <hip/hip_runtime.h>
__global__ void SMWatSSM(float* SSMA, float* SSMB, float* CSM, int M, int N, int diagLen, int diagLenPow2, float hvPenalty, int flip) {
    //Have circularly rotating system of 3 buffers
    extern __shared__ float x[]; //Circular buffer
    int off = 0;
    int upoff = 0;

    //Other local variables
    int i, k;
    int i1, i2, j1, j2;
    int thisi, thisj;
    int idx;
    float val, score;
    int ci = blockIdx.x;
    int cj = blockIdx.y;
    int finished = 0;


    //Figure out K (number of batches)
    int K = diagLenPow2 >> 9;
    if (K == 0) {
        K = 1;
    }

    //Initialize all buffer elements to -1
    for (k = 0; k < K; k++) {
        for (off = 0; off < 3; off++) {
            if (512*k + threadIdx.x < diagLen) {
                x[512*k + threadIdx.x + off*diagLen] = -1;
            }
        }
    }
    off = 0;

    //Process each diagonal
    for (i = 0; i < N + M - 1; i++) {
        if (finished == 1) {
            break;
        }
        //Figure out the bounds of this diagonal
        i1 = i;
        j1 = 0;
        upoff = -1;
        if (i1 >= M) {
            i1 = M-1;
            j1 = i - (M-1);
            upoff = 0;
        }
        j2 = i;
        i2 = 0;
        if (j2 >= N) {
            j2 = N-1;
            i2 = i - (N-1);
        }
        //Update each batch
        for (k = 0; k < K; k++) {
            idx = k*512 + threadIdx.x;
            if (idx >= diagLen) {
                break;
            }
            thisi = i1 - idx;
            thisj = j1 + idx;
            if (thisi < i2 || thisj > j2) {
                x[off*diagLen + idx] = -1;
                continue;
            }
            if (flip) {
                val = SSMA[(M-ci-1)*M + (M-thisi-1)] - SSMB[(N-cj-1)*N + N-thisj-1];
            }
            else {
                val = SSMA[ci*M + thisi] - SSMB[cj*N + thisj];
            }
            if (val < 0) {
                val = val*-1.0f;
            }
            val = expf(-val/0.09f)-0.6f;
            score = 0.0;
            //Above
            if (idx + upoff + 1 < N + M - 1 && thisi > 0) {
                if (x[((off+1)%3)*diagLen + idx + upoff + 1] > -1) {
                    if (val + x[((off+1)%3)*diagLen + idx + upoff + 1] + hvPenalty > score)
                    score = val + x[((off+1)%3)*diagLen + idx + upoff + 1] + hvPenalty;
                }
                else if (val + hvPenalty > score) {
                    score = val + hvPenalty;
                }
                //U[thisi*N + thisj] = x[((off+1)%3)*diagLen + idx + upoff + 1];
            }
            else if (val + hvPenalty > score) {
                score = val + hvPenalty;
            }


            if (idx + upoff >= 0 && thisj > 0) {
                //Left
                if (x[((off+1)%3)*diagLen + idx + upoff] > -1) {
                    if (x[((off+1)%3)*diagLen + idx + upoff] + val + hvPenalty > score) {
                        score = x[((off+1)%3)*diagLen + idx + upoff] + val + hvPenalty;
                    }
                    else if (val + hvPenalty > score) {
                        score = val + hvPenalty;
                    }
                    //L[thisi*N + thisj] = x[((off+1)%3)*diagLen + idx + upoff];
                }
            }
            else if (val + hvPenalty > score) {
                score = val + hvPenalty;
            }


            if (i1 == M-1 && j1 > 1) {
                upoff = 1;
            }
            if (idx + upoff >= 0 && thisi > 0) {
                //Diagonal
                if (x[((off+2)%3)*diagLen + idx + upoff] > -1) {
                    if (x[((off+2)%3)*diagLen + idx + upoff] + val > score) {
                        score = x[((off+2)%3)*diagLen + idx + upoff] + val;
                    }
                }
                else if (val > score) {
                    score = val;
                }
                //UL[thisi*N + thisj] = x[((off+2)%3)*diagLen + idx + upoff];
            }
            else if (val > score) {
                score = val;
            }
            x[off*diagLen + idx] = score;
            if (thisi == ci && thisj == cj) {
                CSM[ci*N + cj] = score;
                finished = 1;
            }
        }
        off = (off + 2) % 3; //Cycle buffers
        __syncthreads();
    }
}
