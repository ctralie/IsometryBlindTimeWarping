
#include <hip/hip_runtime.h>
/*
How much of a difference would it make if I stored the buffers in global memory?
*/

__global__ void DTW(float* CSM, float* D, int M, int N, int diagLen, int diagLenPow2, float* res) {
    //Have circularly rotating system of 3 buffers
    extern __shared__ float x[]; //Circular buffer
    int off = 0;

    //Other local variables
    int i, k;
    int i1, i2, j1, j2;
    int thisi, thisj;
    int idx;
    float val, score;

    //Figure out K (number of batches)
    int K = diagLenPow2 >> 9;
    if (K == 0) {
        K = 1;
    }

    //Initialize all buffer elements to -1
    for (k = 0; k < K; k++) {
        for (off = 0; off < 3; off++) {
            x[512*k + threadIdx.x + off*diagLen] = -1;
        }
    }
    off = 0;

    //Process each diagonal
    for (i = 0; i < N + M - 1; i++) {
        //Figure out the bounds of this diagonal
        i1 = i;
        j1 = 0;
        if (i1 >= M) {
            i1 = M-1;
            j1 = i - (M-1);
        }
        j2 = i;
        i2 = 0;
        if (j2 >= N) {
            j2 = N-1;
            i2 = i - (N-1);
        }
        //Update each batch
        for (k = 0; k < K; k++) {
            idx = k*512 + threadIdx.x;
            if (idx >= diagLen) {
                break;
            }
            thisi = i1 - idx;
            thisj = j1 + idx;
            if (thisi < i2 || thisj > j2) {
                x[off*diagLen + idx] = -1;
                continue;
            }
            val = CSM[thisi*N + thisj];
            score = -1;
            //Above
            if (x[((off+1)%3)*diagLen + idx] > -1) {
                score = val + x[((off+1)%3)*diagLen + threadIdx.x];
            }
            if (idx > 0) {
                //Left
                if (x[((off+1)%3)*diagLen + idx - 1] > -1) {
                    if (score == -1 || x[((off+1)%3)*diagLen + idx - 1] + val < score) {
                        score = x[((off+1)%3)*diagLen + idx - 1] + val;
                    }
                }
                //Diagonal
                if (x[((off+2)%3)*diagLen + idx - 1] > -1) {
                    if (score == -1 || x[((off+2)%3)*diagLen + idx - 1] + val < score) {
                        score = x[((off+2)%3)*diagLen + idx - 1] + val;
                    }
                }
            }
            if (score == -1) {
                score = val;
            }
            D[thisi*N + thisj] = score;
            x[off*diagLen + threadIdx.x] = score;
            if (i == N + M - 2) {
                res[0] = score;
            }
        }
        off = (off + 4) % 3; //Cycle buffers
        __syncthreads();
    }
}
