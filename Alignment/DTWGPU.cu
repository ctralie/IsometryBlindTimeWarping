
#include <hip/hip_runtime.h>
/*
How much of a difference would it make if I stored the buffers in global memory?
*/

//__global__ void DTW(float* CSM, float* D, float* U, float* L, float* UL, int M, int N, int diagLen, int diagLenPow2, float* res) {

__global__ void DTW(float* CSM, int M, int N, int ci, int cj, int diagLen, int diagLenPow2, float* res) {
    //Have circularly rotating system of 3 buffers
    extern __shared__ float x[]; //Circular buffer
    int off = 0;
    int upoff = 0;

    //Other local variables
    int i, k;
    int i1, i2, j1, j2;
    int thisi, thisj;
    int idx;
    float val, score;

    //Figure out K (number of batches)
    int K = diagLenPow2 >> 9;
    if (K == 0) {
        K = 1;
    }

    //Initialize all buffer elements to -1
    for (k = 0; k < K; k++) {
        for (off = 0; off < 3; off++) {
            if (512*k + threadIdx.x < diagLen) {
                x[512*k + threadIdx.x + off*diagLen] = -1;
            }
        }
    }
    off = 0;

    //Process each diagonal
    for (i = 0; i < N + M - 1; i++) {
        //Figure out the bounds of this diagonal
        i1 = i;
        j1 = 0;
        upoff = -1;
        if (i1 >= M) {
            i1 = M-1;
            j1 = i - (M-1);
            upoff = 0;
        }
        j2 = i;
        i2 = 0;
        if (j2 >= N) {
            j2 = N-1;
            i2 = i - (N-1);
        }
        //Update each batch
        for (k = 0; k < K; k++) {
            idx = k*512 + threadIdx.x;
            if (idx >= diagLen) {
                break;
            }
            thisi = i1 - idx;
            thisj = j1 + idx;
            if (thisi < i2 || thisj > j2) {
                x[off*diagLen + idx] = -1;
                continue;
            }
            if (!((thisi <= ci && thisj <= cj) || (thisi >= ci && thisj >= cj))) {
                x[off*diagLen + idx] = -1;
                continue;
            }
            val = CSM[thisi*N + thisj];
            score = -1;
            //Above
            if (idx + upoff + 1 < N + M - 1 && thisi > 0) {
                if (x[((off+1)%3)*diagLen + idx + upoff + 1] > -1) {
                    score = val + x[((off+1)%3)*diagLen + idx + upoff + 1];
                }
                //U[thisi*N + thisj] = x[((off+1)%3)*diagLen + idx + upoff + 1];
            }
            if (idx + upoff >= 0 && thisj > 0) {
                //Left
                if (x[((off+1)%3)*diagLen + idx + upoff] > -1) {
                    if (score == -1 || x[((off+1)%3)*diagLen + idx + upoff] + val < score) {
                        score = x[((off+1)%3)*diagLen + idx + upoff] + val;
                    }
                }
                //L[thisi*N + thisj] = x[((off+1)%3)*diagLen + idx + upoff];
            }
            if (i1 == M-1 && j1 > 1) {
                upoff = 1;
            }
            if (!((thisi == ci && thisj == cj + 1) || (thisi == ci + 1 && thisj == cj))) {
                if (idx + upoff >= 0 && thisi > 0) {
                    //Diagonal
                    if (x[((off+2)%3)*diagLen + idx + upoff] > -1) {
                        if (score == -1 || x[((off+2)%3)*diagLen + idx + upoff] + val < score) {
                            score = x[((off+2)%3)*diagLen + idx + upoff] + val;
                        }
                    }
                    //UL[thisi*N + thisj] = x[((off+2)%3)*diagLen + idx + upoff];
                }
            }
            if (score == -1) {
                score = val;
            }
            x[off*diagLen + idx] = score;
            if (i == N + M - 2) {
                res[0] = score;
            }
        }
        off = (off + 2) % 3; //Cycle buffers
        __syncthreads();
    }
}
